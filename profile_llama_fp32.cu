/*
This code is a convenience tool for profiling the CUDA kernels in the training
loop of train_llama_fp32.cu. Compile:

make profile_llama_fp32cu NO_MULTI_GPU=1

And then e.g. use ncu from NVIDIA. The CLI docs for example:
https://docs.nvidia.com/nsight-compute/NsightComputeCli/

TLDR run like:

sudo ncu --set full --import-source yes -o profile -f ./profile_llama_fp32cu

This:
- `--set full` means we'll collect A LOT of metrics. take out for less
- `--import-source yes` means we'll get the source code in the profile
- `-o profile` writes the results into file profile.ncu-rep
- `-f` forces overwrite of the profile.ncu-rep file
- `./profile_llama_fp32cu` is the executable we want to profile

This writes results into profile.ncu-rep output file.
You can open this up in NVIDIA Nsight Compute UI.
For example, I have NVIDIA Nsight Compute installed on my Mac, and I rsync
the profile.ncu-rep from a cloud box to local to pretty view.
*/

#define TESTING
#include "train_llama_fp32.cu"

int main(int argc, char *argv[])
{
    // Multi-GPU support is not needed, so no multi_gpu_config initialization.

    // Load the LLaMA model parameters
    LLaMA model;
    load_model_params(&model); // Assuming load_model_params loads your model's parameters

    int B = 24;   // if program OOMs decrease this number, e.g. all the way down to 4 or etc
    int T = 1024; // if even that OOMs move on to this one. keep them nice and powers of 2
    printf("batch size: %d\n", B);
    printf("sequence length: %d\n", T);

    int *x = (int *)mallocCheck(B * T * sizeof(int));
    int *y = (int *)mallocCheck(B * T * sizeof(int));
    for (int i = 0; i < B * T; ++i)
    {
        x[i] = i % model.config.vocab_size;
        y[i] = i % model.config.vocab_size;
    }

    // Override number of layers to 1 because all layers repeat the same kernels, only profile once
    model.config.num_layers = 1;

    // Do a training step
    llama3_forward(&model, x, B, T);     // Forward pass
    llama3_backward(&model, x, y, 1, 0); // Backward pass

    // Update model parameters using AdamW optimizer
    llama3_update(&model, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.0f, 1); // Update step

    cudaCheck(hipDeviceSynchronize()); // Finish all CUDA work to get correct precise timings

    // Free resources
    llama_free(&model);
    return 0;
}
